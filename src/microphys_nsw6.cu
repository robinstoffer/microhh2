/*
 * MicroHH
 * Copyright (c) 2011-2020 Chiel van Heerwaarden
 * Copyright (c) 2011-2020 Thijs Heus
 * Copyright (c) 2014-2020 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "fields.h"
#include "thermo.h"
#include "stats.h"
#include "microphys_nsw6.h"

#ifdef USECUDA
template<typename TF>
void Microphys_nsw6<TF>::exec(Thermo<TF>& thermo, const double dt, Stats<TF>& stats)
{
    throw std::runtime_error("Microphys_nsw6 is not implemented yet on the GPU");
}
#endif

#ifdef USECUDA
template<typename TF>
unsigned long Microphys_nsw6<TF>::get_time_limit(unsigned long idt, const double dt)
{
    throw std::runtime_error("Microphys_nsw6 is not implemented yet on the GPU");
}
#endif

template class Microphys_nsw6<double>;
template class Microphys_nsw6<float>;
