#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2020 Chiel van Heerwaarden
 * Copyright (c) 2011-2020 Thijs Heus
 * Copyright (c) 2014-2020 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <cstdio>
#include <iostream>
#include "master.h"
#include "grid.h"
#include "fields.h"
#include "defines.h"
#include "constants.h"
#include "thermo.h"
#include "model.h"
#include "master.h"
#include "tools.h"
#include "timedep.h"
#include "monin_obukhov.h"
#include "boundary_surface.h"

namespace
{
    namespace most = Monin_obukhov;
    const int nzL = 10000; // Size of the lookup table for MO iterations.

    template<typename TF> __device__
    TF find_Obuk_g(const float* const __restrict__ zL, const float* const __restrict__ f,
                       int &n, const TF Ri, const TF zsl)
    {
        // Determine search direction.
        if ((f[n]-Ri) > 0.f)
            while ( (f[n-1]-Ri) > 0.f && n > 0) { --n; }
        else
            while ( (f[n]-Ri) < 0.f && n < (nzL-1) ) { ++n; }

        const TF zL0 = (n == 0 || n == nzL-1) ? zL[n] : zL[n-1] + (Ri-f[n-1]) / (f[n]-f[n-1]) * (zL[n]-zL[n-1]);

        return zsl/zL0;
    }


    template<typename TF> __device__
    TF calc_Obuk_noslip_flux_g(float* __restrict__ zL, float* __restrict__ f, int& n, TF du, TF bfluxbot, TF zsl)
    {
        // Calculate the appropriate Richardson number.
        const TF Ri = -Constants::kappa<TF> * bfluxbot * zsl / pow(du, TF(3));
        return find_Obuk_g(zL, f, n, Ri, zsl);
    }

    template<typename TF> __device__
    TF calc_Obuk_noslip_dirichlet_g(float* __restrict__ zL, float* __restrict__ f, int& n, TF du, TF db, TF zsl)
    {
        // Calculate the appropriate Richardson number.
        const TF Ri = Constants::kappa<TF> * db * zsl / pow(du, TF(2));
        return find_Obuk_g(zL, f, n, Ri, zsl);
    }

    /* Calculate absolute wind speed */
    template<typename TF> __global__
    void du_tot_g(TF* __restrict__ dutot,
                  TF* __restrict__ u,    TF* __restrict__ v,
                  TF* __restrict__ ubot, TF* __restrict__ vbot,
                  int istart, int jstart, int kstart,
                  int iend,   int jend, int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;

        if (i < iend && j < jend)
        {
            const int ii  = 1;
            const int ij  = i + j*jj;
            const int ijk = i + j*jj + kstart*kk;
            const TF minval = 1.e-1;

            const TF du2 = pow(TF(0.5)*(u[ijk] + u[ijk+ii]) - TF(0.5)*(ubot[ij] + ubot[ij+ii]), TF(2))
                         + pow(TF(0.5)*(v[ijk] + v[ijk+jj]) - TF(0.5)*(vbot[ij] + vbot[ij+jj]), TF(2));
            dutot[ij] = fmax(pow(du2, TF(0.5)), minval);
        }
    }

    template<typename TF> __global__
    void stability_g(TF* __restrict__ ustar, TF* __restrict__ obuk,
                     TF* __restrict__ b, TF* __restrict__ bbot, TF* __restrict__ bfluxbot,
                     TF* __restrict__ dutot, float* __restrict__ zL_sl_g, float* __restrict__ f_sl_g,
                     int* __restrict__ nobuk_g,
                     TF z0m, TF z0h, TF db_ref, TF zsl,
                     int icells, int jcells, int kstart, int jj, int kk,
                     Boundary_type mbcbot, Boundary_type thermobc)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        const int j = blockIdx.y*blockDim.y + threadIdx.y;

        if (i < icells && j < jcells)
        {
            const int ij  = i + j*jj;
            const int ijk = i + j*jj + kstart*kk;

            // case 1: fixed buoyancy flux and fixed ustar
            if (mbcbot == Boundary_type::Ustar_type && thermobc == Boundary_type::Flux_type)
            {
                obuk[ij] = -pow(ustar[ij], TF(3)) / (Constants::kappa<TF>*bfluxbot[ij]);
            }
            // case 2: fixed buoyancy flux and free ustar
            else if (mbcbot == Boundary_type::Dirichlet_type && thermobc == Boundary_type::Flux_type)
            {
                obuk [ij] = calc_Obuk_noslip_flux_g(zL_sl_g, f_sl_g, nobuk_g[ij], dutot[ij], bfluxbot[ij], zsl);
                ustar[ij] = dutot[ij] * most::fm(zsl, z0m, obuk[ij]);
            }
            // case 3: fixed buoyancy surface value and free ustar
            else if (mbcbot == Boundary_type::Dirichlet_type && thermobc == Boundary_type::Dirichlet_type)
            {
                TF db = b[ijk] - bbot[ij] + db_ref;
                obuk [ij] = calc_Obuk_noslip_dirichlet_g(zL_sl_g, f_sl_g, nobuk_g[ij], dutot[ij], db, zsl);
                ustar[ij] = dutot[ij] * most::fm(zsl, z0m, obuk[ij]);
            }
        }
    }

    template<typename TF> __global__
    void stability_neutral_g(TF* __restrict__ ustar, TF* __restrict__ obuk,
                             TF* __restrict__ dutot, TF z0m, TF z0h, TF zsl,
                             int icells, int jcells, int kstart, int jj, int kk,
                             Boundary_type mbcbot, Boundary_type thermobc)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        const int j = blockIdx.y*blockDim.y + threadIdx.y;

        if (i < icells && j < jcells)
        {
            const int ij  = i + j*jj;

            // case 1: fixed buoyancy flux and fixed ustar
            if (mbcbot == Boundary_type::Ustar_type && thermobc == Boundary_type::Flux_type)
            {
                obuk[ij] = -Constants::dbig;
            }
            // case 2: fixed buoyancy flux and free ustar
            else if (mbcbot == Boundary_type::Dirichlet_type && thermobc == Boundary_type::Flux_type)
            {
                obuk [ij] = -Constants::dbig;
                ustar[ij] = dutot[ij] * most::fm(zsl, z0m, obuk[ij]);
            }
            // case 3: fixed buoyancy surface value and free ustar
            else if (mbcbot == Boundary_type::Dirichlet_type && thermobc == Boundary_type::Dirichlet_type)
            {
                obuk [ij] = -Constants::dbig;
                ustar[ij] = dutot[ij] * most::fm(zsl, z0m, obuk[ij]);
            }
        }
    }

    template<typename TF> __global__
    void surfm_flux_g(TF* __restrict__ ufluxbot, TF* __restrict__ vfluxbot,
                      TF* __restrict__ u,        TF* __restrict__ v,
                      TF* __restrict__ ubot,     TF* __restrict__ vbot,
                      TF* __restrict__ ustar,    TF* __restrict__ obuk,
                      TF zsl, TF z0m,
                      int istart, int jstart, int kstart,
                      int iend,   int jend, int jj, int kk,
                      Boundary_type bcbot)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;

        if (i < iend && j < jend)
        {
            const int ii  = 1;
            const int ij  = i + j*jj;
            const int ijk = i + j*jj + kstart*kk;

            if (bcbot == Boundary_type::Dirichlet_type)
            {
                // interpolate the whole stability function rather than ustar or obuk
                ufluxbot[ij] = -(u[ijk]-ubot[ij])*TF(0.5)*(ustar[ij-ii]*most::fm(zsl, z0m, obuk[ij-ii]) + ustar[ij]*most::fm(zsl, z0m, obuk[ij]));
                vfluxbot[ij] = -(v[ijk]-vbot[ij])*TF(0.5)*(ustar[ij-jj]*most::fm(zsl, z0m, obuk[ij-jj]) + ustar[ij]*most::fm(zsl, z0m, obuk[ij]));
            }
            else if (bcbot == Boundary_type::Ustar_type)
            {
                const TF minval = 1.e-2;

                // minimize the wind at 0.01, thus the wind speed squared at 0.0001
                const TF vonu2 = fmax(minval, TF(0.25)*( pow(v[ijk-ii]-vbot[ij-ii], TF(2)) + pow(v[ijk-ii+jj]-vbot[ij-ii+jj], TF(2))
                                                       + pow(v[ijk   ]-vbot[ij   ], TF(2)) + pow(v[ijk   +jj]-vbot[ij   +jj], TF(2))) );
                const TF uonv2 = fmax(minval, TF(0.25)*( pow(u[ijk-jj]-ubot[ij-jj], TF(2)) + pow(u[ijk+ii-jj]-ubot[ij+ii-jj], TF(2))
                                                       + pow(u[ijk   ]-ubot[ij   ], TF(2)) + pow(u[ijk+ii   ]-ubot[ij+ii   ], TF(2))) );

                const TF u2 = fmax(minval, pow(u[ijk]-ubot[ij], TF(2)));
                const TF v2 = fmax(minval, pow(v[ijk]-vbot[ij], TF(2)));

                const TF ustaronu4 = TF(0.5)*(pow(ustar[ij-ii], TF(4)) + pow(ustar[ij], TF(4)));
                const TF ustaronv4 = TF(0.5)*(pow(ustar[ij-jj], TF(4)) + pow(ustar[ij], TF(4)));

                ufluxbot[ij] = -copysign(TF(1.), u[ijk]-ubot[ij]) * pow(ustaronu4 / (TF(1.) + vonu2 / u2), TF(0.5));
                vfluxbot[ij] = -copysign(TF(1.), v[ijk]-vbot[ij]) * pow(ustaronv4 / (TF(1.) + uonv2 / v2), TF(0.5));
            }
        }
    }

    template<typename TF> __global__
    void surfm_grad_g(TF* __restrict__ ugradbot, TF* __restrict__ vgradbot,
                      TF* __restrict__ u,        TF* __restrict__ v,
                      TF* __restrict__ ubot,     TF* __restrict__ vbot, TF zsl,
                      int icells, int jcells, int kstart, int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        const int j = blockIdx.y*blockDim.y + threadIdx.y;

        if (i < icells && j < jcells)
        {
            const int ij  = i + j*jj;
            const int ijk = i + j*jj + kstart*kk;

            ugradbot[ij] = (u[ijk]-ubot[ij])/zsl;
            vgradbot[ij] = (v[ijk]-vbot[ij])/zsl;
        }
    }

    template<typename TF> __global__
    void surfs_g(TF* __restrict__ varfluxbot, TF* __restrict__ vargradbot,
                 TF* __restrict__ varbot,     TF* __restrict__ var,
                 TF* __restrict__ ustar,      TF* __restrict__ obuk, TF zsl, TF z0h,
                 int icells, int jcells, int kstart,
                 int jj, int kk,
                 Boundary_type bcbot)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        const int j = blockIdx.y*blockDim.y + threadIdx.y;

        if (i < icells && j < jcells)
        {
            const int ij  = i + j*jj;
            const int ijk = i + j*jj + kstart*kk;

            if (bcbot == Boundary_type::Dirichlet_type)
            {
                varfluxbot[ij] = -(var[ijk]-varbot[ij])*ustar[ij]*most::fh(zsl, z0h, obuk[ij]);
                vargradbot[ij] = (var[ijk]-varbot[ij])/zsl;
            }
            else if (bcbot == Boundary_type::Flux_type)
            {
                varbot[ij]     = varfluxbot[ij] / (ustar[ij]*most::fh(zsl, z0h, obuk[ij])) + var[ijk];
                vargradbot[ij] = (var[ijk]-varbot[ij])/zsl;
            }
        }
    }
}

template<typename TF>
void Boundary_surface<TF>::prepare_device()
{
    auto& gd = grid.get_grid_data();

    const int dmemsize2d = gd.ijcells*sizeof(TF);
    const int imemsize2d = gd.ijcells*sizeof(int);
    const int dimemsize  = gd.icells*sizeof(TF);
    const int iimemsize  = gd.icells*sizeof(int);

    cuda_safe_call(hipMalloc(&obuk_g,  dmemsize2d));
    cuda_safe_call(hipMalloc(&ustar_g, dmemsize2d));
    cuda_safe_call(hipMalloc(&nobuk_g, imemsize2d));

    cuda_safe_call(hipMalloc(&zL_sl_g, nzL*sizeof(float)));
    cuda_safe_call(hipMalloc(&f_sl_g,  nzL*sizeof(float)));

    cuda_safe_call(hipMemcpy2D(obuk_g,  dimemsize, obuk.data(),  dimemsize, dimemsize, gd.jcells, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy2D(ustar_g, dimemsize, ustar.data(), dimemsize, dimemsize, gd.jcells, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy2D(nobuk_g, iimemsize, nobuk.data(), iimemsize, iimemsize, gd.jcells, hipMemcpyHostToDevice));

    cuda_safe_call(hipMemcpy(zL_sl_g, zL_sl.data(), nzL*sizeof(float), hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(f_sl_g,  f_sl.data(),  nzL*sizeof(float), hipMemcpyHostToDevice));
}

// TMP BVS
template<typename TF>
void Boundary_surface<TF>::forward_device()
{
    auto& gd = grid.get_grid_data();

    const int dimemsize   = gd.icells  * sizeof(TF);
    const int iimemsize   = gd.icells  * sizeof(int);

    cuda_safe_call(hipMemcpy2D(obuk_g,  dimemsize, obuk.data(),  dimemsize, dimemsize, gd.jcells, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy2D(ustar_g, dimemsize, ustar.data(), dimemsize, dimemsize, gd.jcells, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy2D(nobuk_g, iimemsize, nobuk.data(), iimemsize, iimemsize, gd.jcells, hipMemcpyHostToDevice));
}

// TMP BVS
template<typename TF>
void Boundary_surface<TF>::backward_device()
{
    auto& gd = grid.get_grid_data();

    const int dimemsize = gd.icells * sizeof(TF);
    const int iimemsize = gd.icells * sizeof(int);

    cuda_safe_call(hipMemcpy2D(obuk.data(),  dimemsize, obuk_g,  dimemsize, dimemsize, gd.jcells, hipMemcpyDeviceToHost));
    cuda_safe_call(hipMemcpy2D(ustar.data(), dimemsize, ustar_g, dimemsize, dimemsize, gd.jcells, hipMemcpyDeviceToHost));
    cuda_safe_call(hipMemcpy2D(nobuk.data(), iimemsize, nobuk_g, iimemsize, iimemsize, gd.jcells, hipMemcpyDeviceToHost));
}

template<typename TF>
void Boundary_surface<TF>::clear_device()
{
    cuda_safe_call(hipFree(obuk_g ));
    cuda_safe_call(hipFree(ustar_g));
    cuda_safe_call(hipFree(nobuk_g));
    cuda_safe_call(hipFree(zL_sl_g));
    cuda_safe_call(hipFree(f_sl_g ));
}

#ifdef USECUDA
template<typename TF>
void Boundary_surface<TF>::update_bcs(Thermo<TF>& thermo)
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;

    // For 2D field excluding ghost cells
    int gridi = gd.imax/blocki + (gd.imax%blocki > 0);
    int gridj = gd.jmax/blockj + (gd.jmax%blockj > 0);
    dim3 gridGPU (gridi,  gridj,  1);
    dim3 blockGPU(blocki, blockj, 1);

    // For 2D field including ghost cells
    gridi = gd.icells/blocki + (gd.icells%blocki > 0);
    gridj = gd.jcells/blockj + (gd.jcells%blockj > 0);
    dim3 gridGPU2 (gridi,  gridj,  1);
    dim3 blockGPU2(blocki, blockj, 1);

    // Calculate dutot in tmp2
    auto dutot = fields.get_tmp_g();

    du_tot_g<<<gridGPU, blockGPU>>>(
        dutot->fld_g,
        fields.mp.at("u")->fld_g,     fields.mp.at("v")->fld_g,
        fields.mp.at("u")->fld_bot_g, fields.mp.at("v")->fld_bot_g,
        gd.istart, gd.jstart, gd.kstart,
        gd.iend, gd.jend, gd.icells, gd.ijcells);
    cuda_check_error();

    // 2D cyclic boundaries on dutot
    boundary_cyclic.exec_2d_g(dutot->fld_g);

    // start with retrieving the stability information
    if (thermo.get_switch() == "0")
    {
        // Calculate ustar and Obukhov length, including ghost cells
        stability_neutral_g<<<gridGPU2, blockGPU2>>>(
            ustar_g, obuk_g,
            dutot->fld_g, z0m, z0h, gd.z[gd.kstart],
            gd.icells, gd.jcells, gd.kstart, gd.icells, gd.ijcells, mbcbot, thermobc);
        cuda_check_error();
    }
    else
    {
        auto buoy = fields.get_tmp_g();
        thermo.get_buoyancy_surf_g(*buoy);
        const TF db_ref = thermo.get_db_ref();

        // Calculate ustar and Obukhov length, including ghost cells
        stability_g<<<gridGPU2, blockGPU2>>>(
            ustar_g, obuk_g,
            buoy->fld_g, buoy->fld_bot_g, buoy->flux_bot_g,
            dutot->fld_g, zL_sl_g, f_sl_g,
            nobuk_g,
            z0m, z0h, db_ref, gd.z[gd.kstart],
            gd.icells, gd.jcells, gd.kstart, gd.icells, gd.ijcells,
            mbcbot, thermobc);
        cuda_check_error();

        fields.release_tmp_g(buoy);
    }

    fields.release_tmp_g(dutot);

    // Calculate surface momentum fluxes, excluding ghost cells
    surfm_flux_g<<<gridGPU, blockGPU>>>(
        fields.mp.at("u")->flux_bot_g, fields.mp.at("v")->flux_bot_g,
        fields.mp.at("u")->fld_g,      fields.mp.at("v")->fld_g,
        fields.mp.at("u")->fld_bot_g,  fields.mp.at("v")->fld_bot_g,
        ustar_g, obuk_g, gd.z[gd.kstart], z0m,
        gd.istart, gd.jstart, gd.kstart,
        gd.iend, gd.jend, gd.icells, gd.ijcells, mbcbot);
    cuda_check_error();

    // 2D cyclic boundaries on the surface fluxes
    boundary_cyclic.exec_2d_g(fields.mp.at("u")->flux_bot_g);
    boundary_cyclic.exec_2d_g(fields.mp.at("v")->flux_bot_g);

    // Calculate surface gradients, including ghost cells
    surfm_grad_g<<<gridGPU2, blockGPU2>>>(
        fields.mp.at("u")->grad_bot_g, fields.mp.at("v")->grad_bot_g,
        fields.mp.at("u")->fld_g,      fields.mp.at("v")->fld_g,
        fields.mp.at("u")->fld_bot_g,  fields.mp.at("v")->fld_bot_g,
        gd.z[gd.kstart], gd.icells, gd.jcells, gd.kstart, gd.icells, gd.ijcells);
    cuda_check_error();

    // Calculate scalar fluxes, gradients and/or values, including ghost cells
    for (auto it : fields.sp)
        surfs_g<<<gridGPU2, blockGPU2>>>(
            it.second->flux_bot_g, it.second->grad_bot_g,
            it.second->fld_bot_g,  it.second->fld_g,
            ustar_g, obuk_g, gd.z[gd.kstart], z0h,
            gd.icells,  gd.jcells, gd.kstart,
            gd.icells, gd.ijcells, sbc.at(it.first).bcbot);
    cuda_check_error();
}
#endif

template class Boundary_surface<double>;
template class Boundary_surface<float>;
